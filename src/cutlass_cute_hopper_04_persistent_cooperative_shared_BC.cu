#include "hip/hip_runtime.h"
#include "cuda_gemm.hpp"

#include <cute/tensor.hpp>
#include <cutlass/cluster_launch.hpp>
#include <cute/arch/copy_sm90.hpp>
#include <cutlass/arch/barrier.h>
#include <cutlass/pipeline/pipeline.hpp>
#include <cutlass/arch/reg_reconfig.h>

#include "cutlass/fast_math.h"

using namespace cute;

namespace gemm_hopper_v04
{
// shared storage
template <
    typename T,
    int PIPE,
    typename SmemLayoutA,
    typename SmemLayoutB,
    typename SmemLayoutC
>
struct SharedStorage
{
    // data storage
    array_aligned<T, cosize_v<SmemLayoutA>, 128> smem_A;

    union
    {
        array_aligned<T, cosize_v<SmemLayoutB>, 128> smem_B;
        array_aligned<T, cosize_v<SmemLayoutC>, 128> smem_C;
    }

    // pipeline
    typename cutlass::PipelineTmaAsync<PIPE>::SharedStorage pipeline_A;
    typename cutlass::PipelineTmaAsync<PIPE>::SharedStorage pipeline_B;
    // barrier
    typename cutlass::CutlassBarrier barrier_C;

    int tile_count_semaphore;
};

// kernel traits
template <
    typename T,
    int kWarps_,
    int kBlockM_,
    int kBlockN_,
    int kBlockK_,
    int kStages_
>
struct KernelTraits
{
    using Element = T;
    
    static constexpr int kWarps = kWarps_;
    static_assert(kWarps == 12, "Only support 12 warps now");
    static constexpr int kWarpGroups = kWarps / 4;
    static constexpr int kConsumerWGs = kWarpGroups - 1;
    static constexpr int kThreads = kWarps * 32;

    static constexpr int kBlockM = kBlockM_;
    static constexpr int kBlockN = kBlockN_;
    static constexpr int kBlockK = kBlockK_;
    static constexpr int kStages = kStages_;

    // TiledMMA
    // using mma_op = decltype(
    //     GMMA::ss_op_selector <
    //         Element, Element, Element,
    //         Shape<Int<kBlockM>, Int<kBlockN>, Int<kBlockK>>
    //     >()
    // );
    using mma_op = decltype(
        SM90_64x64x16_F16F16F16_SS<GMMA::Major::K,GMMA::Major::K>{}
    );
    using mma_traits = MMA_Traits<mma_op>;
    using mma_atom = MMA_Atom<mma_traits>;
    
    // thread repetition
    static constexpr int kMmaEURepeatM = kConsumerWGs;
    static constexpr int kMmaEURepeatN = 1;
    static constexpr int kMmaEURepeatK = 1;

    // thread workload repetition
    using mma_atom_shape = mma_traits::Shape_MNK;
    static constexpr int MmaVM = 1 * kMmaEURepeatM * get<0>(mma_atom_shape{});
    static constexpr int MmaVN = 1 * kMmaEURepeatN * get<1>(mma_atom_shape{});
    static constexpr int MmaVK = 1 * kMmaEURepeatK * get<2>(mma_atom_shape{});
    // this is for problem shape (64x1x1) x (64x1x1) x (16x1x1) = 64x64x16
    
    // Thread repetition 1x1x1 --> 128 threads
    using MMA_EU_RepeatT = decltype(
        make_layout(
            make_shape(Int<kMmaEURepeatM>{}, Int<kMmaEURepeatN>{}, Int<kMmaEURepeatK>{})
        )
    );

    // Thread workload repetition 1x1x1
    // Each mode of this shape can be a layout to do permutation on the corresponding layout mode
    using MMA_V_RepeatT = decltype(
        make_shape(Int<MmaVM>{}, Int<MmaVN>{}, Int<MmaVK>{})
    );

    using TiledMMA = decltype(
        make_tiled_mma(
            mma_atom{},
            MMA_EU_RepeatT{},
            MMA_V_RepeatT{}
        )
    );

    using SmemLayoutAtom = GMMA::Layout_K_SW128_Atom<T>;

    using SmemLayoutA = decltype(
        tile_to_shape(
            SmemLayoutAtom{},
            make_shape(Int<kBlockM>{}, Int<kBlockK>{}, Int<kStages>{})
        )
    );

    using SmemLayoutB = decltype(
        tile_to_shape(
            SmemLayoutAtom{},
            make_shape(Int<kBlockN>{}, Int<kBlockK>{}, Int<kStages>{})
        )
    );

    // Rmem to Smem CopyAtom and Layout
    // using SmemCopyAtomC = Copy_Atom<SM90_U32x4_STSM_N, Element>;
    using SmemCopyAtomC = Copy_Atom<SM90_U16x8_STSM_T, Element>;

    // using SmemLayoutAtomC = GMMA::Layout_K_SW128_Atom<T>;
    using SmemLayoutAtomC = GMMA::Layout_MN_SW128_Atom<T>;
    using SmemLayoutC = decltype(
        tile_to_shape(
            SmemLayoutAtomC{},
            make_shape(Int<kBlockM>{}, Int<kBlockN>{})
        )
    );

    // SharedStorage
    using SharedStorage = SharedStorage<T, kStages, SmemLayoutA, SmemLayoutB, SmemLayoutC>;

    // smem_size
    static constexpr int smem_size = sizeof(SharedStorage);

    // MainloopPipeline
    using MainloopPipeline = cutlass::PipelineTmaAsync<kStages>;
};

// collective mainloop
template <typename Kernel_traits>
struct CollectiveMainloop
{
    // 1. extract Kernel_traits
    using Element = typename Kernel_traits::Element;

    static constexpr int kBlockM = Kernel_traits::kBlockM;
    static constexpr int kBlockN = Kernel_traits::kBlockN;
    static constexpr int kBlockK = Kernel_traits::kBlockK;

    using SmemLayoutA = typename Kernel_traits::SmemLayoutA;
    using SmemLayoutB = typename Kernel_traits::SmemLayoutB;

    using TiledMMA = typename Kernel_traits::TiledMMA;

    // 2. decltype of TMA desc
    using ShapeT = Shape<int32_t, int32_t>;
    using StrideT = Shape<int32_t, _1>;
    using LayoutT = Layout<ShapeT, StrideT>;

    using TmaLoadA = decltype(
        make_tma_copy(
            SM90_TMA_LOAD{},
            make_tensor(
                make_gmem_ptr(static_cast<Element *>(nullptr)),
                ShapeT{},
                StrideT{}
            ),
            SmemLayoutA{}(_, _, 0)
        )
    );
    using TmaLoadB = decltype(
        make_tma_copy(
            SM90_TMA_LOAD{},
            make_tensor(
                make_gmem_ptr(static_cast<Element *>(nullptr)),
                ShapeT{},
                StrideT{}
            ),
            SmemLayoutB{}(_, _, 0)
        )
    );

    // 3. set the TMA transaction bytes
    static constexpr int kTmaTransactionBytesA = sizeof(ArrayEngine<Element, size(SmemLayoutA{}(_, _, 0))>);
    static constexpr int kTmaTransactionBytesB = sizeof(ArrayEngine<Element, size(SmemLayoutB{}(_, _, 0))>);
    static constexpr int kTmaTransactionBytes = kTmaTransactionBytesA + kTmaTransactionBytesB; // must be a multiple of 16

    // 4. setup the TMA desc (which replace the need of Params in hopper_00)
    // Host-side kernel arguments
    struct Arguments
    {
        Element const* A;
        Element const* B;
        LayoutT gmemLayoutA;
        LayoutT gmemLayoutB;
    };

    // Device-side kernel params
    struct Params
    {
        LayoutT gmemLayoutA;
        LayoutT gmemLayoutB;
        TmaLoadA tma_load_A;
        TmaLoadB tma_load_B;
    };

    static Params
    to_underlying_arguments(const Arguments& args)
    {
        Tensor mA = make_tensor(
            make_gmem_ptr(args.A),
            args.gmemLayoutA
        );

        Tensor mB = make_tensor(
            make_gmem_ptr(args.B),
            args.gmemLayoutB
        );

        TmaLoadA tma_load_A = make_tma_copy(
            SM90_TMA_LOAD{},
            mA,
            SmemLayoutA{}(_, _, 0)
        );
        TmaLoadB tma_load_B = make_tma_copy(
            SM90_TMA_LOAD{},
            mB,
            SmemLayoutB{}(_, _, 0)
        );

        return
        {
            args.gmemLayoutA,
            args.gmemLayoutB,
            tma_load_A,
            tma_load_B
        };
    }

    // 5. prefetch TMA desc
    CUTLASS_DEVICE
    static void prefetch_tma_descriptors(Params const& mainloop_params)
    {
        cute::prefetch_tma_descriptor(mainloop_params.tma_load_A.get_tma_descriptor());
        cute::prefetch_tma_descriptor(mainloop_params.tma_load_B.get_tma_descriptor());
    }

    // 6. producer
    using MainloopPipeline = typename Kernel_traits::MainloopPipeline;
    using PipelineState = typename MainloopPipeline::PipelineState;

    template <typename SharedStorage>
    CUTLASS_DEVICE
    static void load
    (
        Params const& mainloop_params,
        MainloopPipeline pipeline,
        PipelineState& write_state,
        SharedStorage& shared_storage,
        cute::tuple<int32_t, int32_t, int32_t> block_coord,
        int NUM_TILES_K
    )
    {
        auto [m_block, n_block, _] = block_coord;

        // gmem tensors
        Tensor mA = mainloop_params.tma_load_A.get_tma_tensor(mainloop_params.gmemLayoutA.shape());
        Tensor mB = mainloop_params.tma_load_B.get_tma_tensor(mainloop_params.gmemLayoutB.shape());

        // tiling
        auto cta_tiler = make_shape(Int<kBlockM>{}, Int<kBlockN>{}, Int<kBlockK>{});
        auto cta_coord = make_coord(m_block, n_block, _);
        Tensor gA = local_tile(
            mA,
            cta_tiler,
            cta_coord,
            Step<_1, X, _1>{}
        );  // kBlockM x kBlockK x NUM_TILES_K
        Tensor gB = local_tile(
            mB,
            cta_tiler,
            cta_coord,
            Step<X, _1, _1>{}
        );  // kBlockN x kBlockK x NUM_TILES_K

        // smem tensors
        Tensor sA = make_tensor(
            make_smem_ptr(shared_storage.smem_A.data()),
            SmemLayoutA{}
        );  // kBlockM x kBlockK x PIPE
        Tensor sB = make_tensor(
            make_smem_ptr(shared_storage.smem_B.data()),
            SmemLayoutB{}
        ); // kBlockN x kBlockK x PIPE

        // copy partition
        auto [tAgA, tAsA] = tma_partition(
            mainloop_params.tma_load_A,
            _0{}, Layout<_1>{},
            group_modes<0,2>(sA),
            group_modes<0,2>(gA)
        ); // (TMA, NUM_TILES_K) and (TMA, PIPE)
        auto [tBgB, tBsB] = tma_partition(
            mainloop_params.tma_load_B,
            _0{}, Layout<_1>{},
            group_modes<0,2>(sB),
            group_modes<0,2>(gB)
        ); // (TMA, NUM_TILES_K) and (TMA, PIPE)

        int lane_predicate = cute::elect_one_sync();

        // copy
        if (lane_predicate)
        {
            #pragma unroll 1
            for (int k_tile = 0; k_tile < NUM_TILES_K; ++k_tile)
            {
                pipeline.producer_acquire(write_state);
                // empty_barrier.wait()
                // full_barrier.arrive_and_expect_tx()
                uint64_t* full_barrier = pipeline.producer_get_barrier(write_state);

                auto stage = write_state.index();

                copy(mainloop_params.tma_load_A.with(*full_barrier, 0), tAgA(_, k_tile), tAsA(_, stage));
                copy(mainloop_params.tma_load_B.with(*full_barrier, 0), tBgB(_, k_tile), tBsB(_, stage));

                ++write_state;
            }
        }
    }

    // 7. consumers
    template <
        typename SharedStorage,
        typename FragmentTensorC
    >
    CUTLASS_DEVICE
    static void mma
    (
        Params const& mainloop_params,
        MainloopPipeline pipeline,
        PipelineState& read_state,
        FragmentTensorC& tCrC,
        SharedStorage& shared_storage,
        cute::tuple<int32_t, int32_t, int32_t> block_coord,
        int NUM_TILES_K
    )
    {
        auto [m_block, n_block, _] = block_coord;

        // tiling
        auto cta_tiler = make_shape(Int<kBlockM>{}, Int<kBlockN>{}, Int<kBlockK>{});
        auto cta_coord = make_coord(m_block, n_block, _);

        // smem tensors
        Tensor sA = make_tensor(
            make_smem_ptr(shared_storage.smem_A.data()),
            SmemLayoutA{}
        );  // kBlockM x kBlockK x PIPE
        Tensor sB = make_tensor(
            make_smem_ptr(shared_storage.smem_B.data()),
            SmemLayoutB{}
        ); // kBlockN x kBlockK x PIPE

        // partition
        TiledMMA tiled_mma;
        ThrMMA thr_mma = tiled_mma.get_thread_slice(threadIdx.x - cutlass::NumThreadsPerWarpGroup);

        Tensor tCsA = thr_mma.partition_A(sA);  // (MMA,MMA_M,MMA_K, PIPE)
        Tensor tCsB = thr_mma.partition_B(sB);  // (MMA,MMA_N,MMA_K, PIPE)

        // allocate "fragments"
        // note that in Ampere, the fragments are physically stored in registers
        // but in Hopper, the fragments are not, the tCrA and tCrB are actually the iterators not the data, but still in registers
        Tensor tCrA = thr_mma.make_fragment_A(tCsA);  // iterator (1, MMA_M, MMA_K, PIPE)
        Tensor tCrB = thr_mma.make_fragment_B(tCsB);  // iterator (1, MMA_N, MMA_K, PIPE)

         // MAINLOOP MMA
        #pragma unroll 1
        for (int k_tile = 0; k_tile < NUM_TILES_K; ++k_tile) {
            // Wait for TMA to load this stage of the pipeline
            pipeline.consumer_wait(read_state);
            auto stage = read_state.index();
            warpgroup_arrive();
            // WGMMA with dispatch mode (V,M,K) x (V,N,K) => (V,M,N)
            gemm(tiled_mma, tCrC, tCrA(_,_,_,stage), tCrB(_,_,_,stage), tCrC);
            warpgroup_commit_batch();
        
            // Wait for all MMAs in a K_TILE to complete
            warpgroup_wait<0>();

            // Release the stage of the pipeline for TMA
            pipeline.consumer_release(read_state);
            ++read_state;
        }

        // Make sure all warpgroups have finished mma
        cutlass::arch::NamedBarrier::sync(Kernel_traits::kConsumerWGs * 32 * 4, 0);
    }
};

// collective epilogue
template <
    typename Kernel_traits
>
struct CollectiveEpilogue
{
    // 1. extract Kernel_traits
    using Element = typename Kernel_traits::Element;

    static constexpr int kBlockM = Kernel_traits::kBlockM;
    static constexpr int kBlockN = Kernel_traits::kBlockN;
    static constexpr int kBlockK = Kernel_traits::kBlockK;

    using SmemLayoutC = typename Kernel_traits::SmemLayoutC;
    using SmemCopyAtomC = typename Kernel_traits::SmemCopyAtomC; // (Rmem to Smem)

    using TiledMMA = typename Kernel_traits::TiledMMA;

    // 2. decltype of TMA desc (Smem to Gmem)
    using ShapeT = Shape<int32_t, int32_t>;
    using StrideT = Shape<_1, int32_t>;
    // using StrideT = Shape<int32_t, _1>;
    using LayoutT = Layout<ShapeT, StrideT>;

    using TmaStoreC = decltype(
        make_tma_copy(
            SM90_TMA_STORE{},
            make_tensor(
                make_gmem_ptr(static_cast<Element *>(nullptr)),
                ShapeT{},
                StrideT{}
            ),
            SmemLayoutC{}
        )
    );

    // 3. setup the TMA desc (which replace the need of Params in hopper_00)
    // Host-side kernel arguments
    struct Arguments
    {
        Element* C;
        LayoutT gmemLayoutC;
    };

    // Device-side kernel params
    struct Params
    {
        Element *C;
        LayoutT gmemLayoutC;
        TmaStoreC tma_store_C;
    };

    static Params
    to_underlying_arguments(const Arguments& args)
    {
        return {
            args.C,
            args.gmemLayoutC,
            make_tma_copy(
                SM90_TMA_STORE{},
                make_tensor(
                    make_gmem_ptr(args.C),
                    args.gmemLayoutC
                ),
                SmemLayoutC{}
            )
        };
    }

    // 4. Prefetch TMA desc
    CUTLASS_DEVICE
    static void prefetch_tma_descriptors(Params const& mainloop_params)
    {
        cute::prefetch_tma_descriptor(mainloop_params.tma_store_C.get_tma_descriptor());
    }
    
    // 5. Producer store
    template <
        typename SharedStorage,
        typename FragmentTensorC
    >
    CUTLASS_DEVICE
    static void store(
        Params const& epilogue_params,
        FragmentTensorC const& tCrC,
        SharedStorage& shared_storage,
        cute::tuple<int32_t, int32_t, int32_t> block_coord
    )
    {   
        tma_store_wait<0>();

        auto [m_block, n_block, _] = block_coord;

        // Smem Tensors
        Tensor sC = make_tensor(
            make_smem_ptr(shared_storage.smem_C.data()),
            SmemLayoutC{}
        );

        // Rmem -> Smem Partition
        TiledMMA tiled_mma;
        auto r2s_tiled_copy = make_tiled_copy_C(SmemCopyAtomC{}, tiled_mma);
        // Normal TiledCopy need CopyAtom and ThreadLayout and ValueLayout
        // Here the TiledMMA will provide that information
        auto r2s_thr_copy = r2s_tiled_copy.get_thread_slice(threadIdx.x - cutlass::NumThreadsPerWarpGroup);
        
        // R2S partition
        // tCrC (MMA, MMA_M, MMA_N) --> tAccCrC ((Atom, AtomNum), MMA_M, MMA_N)
        Tensor tAccCrC = r2s_thr_copy.retile_S(tCrC); // copy view of tCrC
        Tensor tAccCsC = r2s_thr_copy.partition_D(sC); // ((Atom, AtomNum), PIPE_M, PIPE_N)

        copy(r2s_tiled_copy, tAccCrC, tAccCsC);
        cutlass::arch::fence_view_async_shared(); // ensure smem writes are visible to TMA
        cutlass::arch::NamedBarrier::arrive(Kernel_traits::kConsumerWGs * 32 * 4 + cutlass::NumThreadsPerWarp,
                                            cutlass::arch::ReservedNamedBarriers::EpilogueBarrier);
        // First we have all Consumer threads arrive at the barrier
        // However we wait for additional 32 threads to arrive? Where and why are these 32 threads?
        // These 32 threads is the last warp used for the TMA store (So this warp actually arrives at the barrier twice)

        // Prefetch TMA Store
        // Gmem Tensors
        Tensor mC = epilogue_params.tma_store_C.get_tma_tensor(epilogue_params.gmemLayoutC.shape());
        
        // Tiling
        auto cta_tiler = make_shape(Int<kBlockM>{}, Int<kBlockN>{}, Int<kBlockK>{});
        auto cta_coord = make_coord(m_block, n_block, _);
        Tensor gC = local_tile(
            mC,
            cta_tiler,
            cta_coord,
            Step<_1, _1, X>{}
        );
        auto s2g_thr_copy = epilogue_params.tma_store_C.get_thread_slice(threadIdx.x - cutlass::NumThreadsPerWarpGroup);
        Tensor tCsC = s2g_thr_copy.partition_S(sC); // (TMA, TMA_M, TMA_N)
        Tensor tCgC = s2g_thr_copy.partition_D(gC); // (TMA, TMA_M, TMA_N)

         // TMA STORE: SMEM -> GMEM
        int write_warp_idx = Kernel_traits::kWarps - 1;
        int const warp_idx = cutlass::canonical_warp_idx_sync();
        int const lane_predicate = cute::elect_one_sync();
        if (warp_idx == write_warp_idx) {
            // Ensure RMEM -> SMEM copy completes before issuing TMA store
            cutlass::arch::NamedBarrier::sync(
                Kernel_traits::kConsumerWGs * 32 * 4 + cutlass::NumThreadsPerWarp, 
                cutlass::arch::ReservedNamedBarriers::EpilogueBarrier
            );
        }
        if (warp_idx == write_warp_idx && lane_predicate) {
            copy(epilogue_params.tma_store_C, tCsC, tCgC);
            tma_store_arrive();
        }
        // TODO: overlap epilogue with next CTA load in persistent kernel
        // tma_store_wait<0>();
    }

    CUTLASS_DEVICE 
    static void store_tail() {
        tma_store_wait<0>();
    }

};

class StaticPersistentTileScheduler
{

public:

    // Host-side kernel arguments
    struct Arguments
    {
        int const num_blocks_m, num_blocks_n;
        int* const tile_count_semaphore = nullptr;
    };

    // Device-side kernel params
    struct Params
    {
        int total_blocks;
        cutlass::FastDivmod m_block_divmod, n_block_divmod;
    };

    static Params
    to_underlying_arguments(const Arguments& args)
    {
        return
        {
            args.num_blocks_m * args.num_blocks_n,
            cutlass::FastDivmod(args.num_blocks_m),
            cutlass::FastDivmod(args.num_blocks_n)
        };
    }

    static dim3
    get_grid_dim(const Arguments& args, int num_sm)
    {
        return {uint32_t(num_sm), 1, 1};
    }

    struct WorkTileInfo
    {
        int tile_idx;

        CUTLASS_DEVICE
        bool is_valid(Params const& params) const {
            return tile_idx < params.total_blocks;
        }

        CUTLASS_DEVICE
        cute::tuple<int32_t, int32_t, int32_t>
        get_block_coord(Params const& params) const {
            int m_block, n_block, bidb;
            bidb = params.n_block_divmod.divmod(n_block, params.m_block_divmod.divmod(m_block, tile_idx));
            return {m_block, n_block, bidb};
        }
    };

    CUTLASS_DEVICE // inline
    StaticPersistentTileScheduler(int* tile_count_smem_) {};

    CUTLASS_DEVICE
    WorkTileInfo
    get_initial_work() const
    {
        return {int(blockIdx.x)};
    }

    CUTLASS_DEVICE
    void init_consumer() const
    {

    }

    CUTLASS_DEVICE
    void prefetch_next_work(const Params& params, WorkTileInfo& current_work) const
    {

    }

    CUTLASS_DEVICE
    void broadcast_next_work(WorkTileInfo& current_work) const
    {

    }

    template<bool isProducer=false>
    CUTLASS_DEVICE
    WorkTileInfo
    get_next_work(const Params& params, WorkTileInfo& current_work) const
    {
        return {current_work.tile_idx + int(gridDim.x)};
    }
}

// kernel
template <
    typename Kernel_traits,
    typename TileScheduler
>
__global__ void cute_hopper_gemm_v04(
    CUTE_GRID_CONSTANT typename CollectiveMainloop<Kernel_traits>::Params const mainloop_params,
    CUTE_GRID_CONSTANT typename CollectiveEpilogue<Kernel_traits>::Params const epilogue_params,
    CUTE_GRID_CONSTANT typename TileScheduler::Params const scheduler_params
)
{   
    using CollectiveMainloop = CollectiveMainloop<Kernel_traits>;
    using CollectiveEpilogue = CollectiveEpilogue<Kernel_traits>;

    using MainloopPipeline = typename Kernel_traits::MainloopPipeline;
    using PipelineParams = typename MainloopPipeline::Params;
    using PipelineState = typename MainloopPipeline::PipelineState;
    // The Synchronization is orchestrated by the pipeline + pipeline state
    // each thread has its own pipeline state to control the synchronization
    // pipeline state = phase bit + stage + count
    // phase bit is initialized to 0 and flip between 0 and 1
    // stage increments by 1 each time but reset to 0 when it reaches kStages
    // count increments by 1 each time

    // shared memory for data + pipeline
    using SharedStorage = typename Kernel_traits::SharedStorage;
    extern __shared__ char smem_[];
    auto &shared_storage = *reinterpret_cast<SharedStorage*>(smem_);
   
    // Only one thread is elected to perfrom the prefetch
    int warp_idx = cutlass::canonical_warp_idx_sync();
    int lane_predicate = cute::elect_one_sync();

    // prefetch TMA Descriptor
    if (warp_idx == 0 && lane_predicate)
    {
        CollectiveMainloop::prefetch_tma_descriptors(mainloop_params);
        CollectiveEpilogue::prefetch_tma_descriptors(epilogue_params);
    }

    // barrier initialization
    if (warp_idx == 0 && lane_predicate)
    {
        shared_storage.barrier_C.init(/*numThreads=*/1);
    }
    // pipeline initialization
    PipelineParams pipeline_params;

    // set the transaction size
    // Remember that the transaction size is passed to arrive_and_expect_tx of the barrier
    // However, the pipeline.producer_acquire_tx() will do that for us:
    // 1. empty_barrier.wait()
    // 2. full_barrier.arrive_and_expect_tx() (only Hopper does this 2.)
    pipeline_params.transaction_bytes = CollectiveMainloop::kTmaTransactionBytes;
    
    // set the role
    int warp_group_idx = cutlass::canonical_warp_group_idx();    
    pipeline_params.role = warp_group_idx == 0
        ? MainloopPipeline::ThreadCategory::Producer
        : MainloopPipeline::ThreadCategory::Consumer;

    // set the thread leader (scope of warp_group)
    const int warp_group_thread_idx = threadIdx.x % cutlass::NumThreadsPerWarpGroup;
    pipeline_params.is_leader = warp_group_thread_idx == 0;
    pipeline_params.num_consumers = cutlass::NumThreadsPerWarpGroup * Kernel_traits::kConsumerWGs;
    
    MainloopPipeline pipeline(
        shared_storage.pipeline,    // address of the pipeline
        pipeline_params,
        Shape<_1, _1, _1>{}         // todo
    );

    const int NUM_TILES_K = cutlass::ceil_div(
        shape<1>(mainloop_params.gmemLayoutA),
        Kernel_traits::kBlockK
    );

    // We need this to guarantee that the Pipeline init is visible to all producers and consumer blocks in the Cluster
    // This is similar to have the barrier visible to all threads in the Cluster
    cluster_sync();

    // Producer
    if (warp_group_idx == 0)
    {
        cutlass::arch::warpgroup_reg_dealloc<24>();
        int warp_idx_in_warpgroup = __shfl_sync(
            0xffffffff,                     // mask: all lanes in the warp
            (threadIdx.x / 32) % 4,         // value: warp_idx_in_warpgroup
            0                               // src lane
        );

        // only the first warp in the warp group will do the load
        // and only the elected thread inside the first warp will do the load
        if (warp_idx_in_warpgroup == 0)
        {
            PipelineState write_state = cutlass::make_producer_start_state<MainloopPipeline>();
            // phase bit = 1, stage = 0, count = 0
            // 1. the phase bit of write_state is to control the empty_barrier (sounds counterintuitive):
            // pipeline.producer_acquire(write_state) or empty_barrier.wait(): when all threads have arrived, the phase bit will be flipped to 1
            // 2. the stage of write_state is to control the full_barrier:
            // pipeline.producer_get_barrier(write_state): it gets the full_barrier_ptr_[stage]

            TileScheduler scheduler(&shared_storage.tile_count_semaphore);

            for (
                auto work_tile_info = scheduler.get_initial_work();
                work_tile_info.is_valid(scheduler_params);
                work_tile_info = scheduler.template get_next_work</*isProducer=*/true>(scheduler_params, work_tile_info)
            )
            {
                auto block_coord = work_tile_info.get_block_coord();

                CollectiveMainloop::load(
                    mainloop_params,
                    pipeline,
                    write_state,
                    shared_storage,
                    block_coord,
                    NUM_TILES_K
                );
            }
        }
    }
    else    // Consumer
    {
        cutlass::arch::warpgroup_reg_alloc<240>();
        PipelineState read_state;

        Tensor tCrC = partition_fragment_C(typename Kernel_traits::TiledMMA{}, Shape<Int<Kernel_traits::kBlockM>, Int<Kernel_traits::kBlockN>>{});
        clear(tCrC);

        TileScheduler scheduler(&shared_storage.tile_count_semaphore);

        for (
            auto work_tile_info = scheduler.get_initial_work();
            work_tile_info.is_valid(scheduler_params);
            work_tile_info = scheduler.template get_next_work</*isProducer=*/false>(scheduler_params, work_tile_info)
        )
        {
            auto block_coord = work_tile_info.get_block_coord();

            CollectiveMainloop::mma(
                mainloop_params,
                pipeline,
                read_state,
                tCrC,
                shared_storage,
                block_coord,
                NUM_TILES_K
                );
            
            CollectiveEpilogue::store(
                epilogue_params,
                tCrC,
                shared_storage,
                block_coord
            );
        }

        CollectiveEpilogue::store_tail();
    }
}

// launch
template<typename T>
void launch_cute_hopper_gemm_kernel_v04(
    size_t m, size_t n, size_t k,
    const T *alpha,
    const T *A, size_t lda,
    const T *B, size_t ldb,
    const T *beta,
    T *C, size_t ldc,
    hipStream_t stream
)
{   
    // Block shape and cta tiler
    constexpr int kWarps_ = 12;
    constexpr int kBlockM_ = 128;
    constexpr int kBlockN_ = 128;
    constexpr int kBlockK_ = 64;
    constexpr int kStages_ = 2;

    using Kernel_traits = KernelTraits<T, kWarps_,kBlockM_, kBlockN_, kBlockK_, kStages_>;

    using SmemLayoutA = typename Kernel_traits::SmemLayoutA;
    using SmemLayoutB = typename Kernel_traits::SmemLayoutB;
    using SmemLayoutC = typename Kernel_traits::SmemLayoutC;

    using TiledMMA = typename Kernel_traits::TiledMMA;

    // setup TMA desc like v00 but using CollectiveMainloop
    int M = int(m); int N = int(n); int K = int(k);
    auto gmemLayoutA = make_layout(make_shape(M, K), make_stride(K, _1{}));
    auto gmemLayoutB = make_layout(make_shape(N, K), make_stride(K, _1{}));
    auto gmemLayoutC = make_layout(make_shape(M, N), make_stride(_1{}, M));
    // auto gmemLayoutC = make_layout(make_shape(M, N), make_stride(N, _1{}));

    using Collective_mainloop = CollectiveMainloop<Kernel_traits>;
    using Collective_epilogue = CollectiveEpilogue<Kernel_traits>;

    using Scheduler = StaticPersistentTileScheduler;

    typename Collective_mainloop::Params mainloop_params = Collective_mainloop::to_underlying_arguments(
        {
            A, B,
            gmemLayoutA, gmemLayoutB
        }
    );

    typename Collective_epilogue::Params epilogue_params = Collective_epilogue::to_underlying_arguments(
        {
            C,
            gmemLayoutC
        }  
    );

    int num_block_m = ceil_div(m, kBlockM_);
    int num_block_n = ceil_div(n, kBlockN_);

    typename Scheduler::Arguments scheduler_args = {num_block_m, num_block_n};
    typename Scheduler::Params scheduler_params = Scheduler::to_underlying_arguments(scheduler_args);

    // Launch parameter setup
    constexpr int smem_size = Kernel_traits::smem_size;
    dim3 block{Kernel_traits::kThreads, 1U, 1U};
    dim3 cluster{1, 1, 1};

    int device;
    hipGetDevice(&device);
    int sm_count;
    hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, device);
    dim3 grid = Scheduler::get_grid_dim(scheduler_params, sm_count);
    cutlass::ClusterLaunchParams launch_params{grid, block, cluster, smem_size, stream};

    void const* kernel = reinterpret_cast<void const*>(&cute_hopper_gemm_v04 <Kernel_traits, Scheduler>);

    if (smem_size >= 48 * 1024) // 48KB
    {
        CUTE_CHECK_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));
    }

    // kernel launch
    cutlass::Status status = cutlass::launch_kernel_on_cluster(
        launch_params,
        kernel,
        mainloop_params,
        epilogue_params,
        scheduler_params
    );
    CUTE_CHECK_LAST();

    if (status != cutlass::Status::kSuccess)
    {
        std::cerr << "Kernel launch failed with status: " << std::endl;
    }

}

// explicit instantiation                      
template void launch_cute_hopper_gemm_kernel_v04<cute::half_t>(size_t m, size_t n, size_t k,
                                    const cute::half_t *alpha,
                                    const cute::half_t *A, size_t lda,
                                    const cute::half_t *B, size_t ldb,
                                    const cute::half_t *beta,
                                    cute::half_t *C, size_t ldc,
                                    hipStream_t stream);                                    

} // namespace gemm_hopper_v04