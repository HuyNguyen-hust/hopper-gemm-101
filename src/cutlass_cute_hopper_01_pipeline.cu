#include "hip/hip_runtime.h"
#include "cuda_gemm.hpp"

#include <cute/tensor.hpp>
#include <cutlass/cluster_launch.hpp>
#include <cute/arch/copy_sm90.hpp>
#include <cutlass/arch/barrier.h>
#include <cutlass/pipeline/pipeline.hpp>
#include <cutlass/arch/reg_reconfig.h>

using namespace cute;

// shared storage
template <
    typename T,
    int PIPE,
    typename SmemLayoutA,
    typename SmemLayoutB
>
struct SharedStorage
{
    // data storage
    array_aligned<T, cosize_v<SmemLayoutA>, 128> smem_A;
    array_aligned<T, cosize_v<SmemLayoutB>, 128> smem_B;

    // pipeline
    typename cutlass::PipelineTmaAsync<PIPE>::SharedStorage pipeline;
};

// kernel traits
template <
    typename T,
    int kWarps_,
    int kBlockM_,
    int kBlockN_,
    int kBlockK_,
    int kStages_
>
struct KernelTraits
{
    using Element = T;
    
    static constexpr int kWarps = kWarps_;
    static_assert(kWarps == 12, "Only support 12 warps now");
    static constexpr int kWarpGroups = kWarps / 4;
    static constexpr int kConsumerWGs = kWarpGroups - 1;
    static constexpr int kThreads = kWarps * 32;

    static constexpr int kBlockM = kBlockM_;
    static constexpr int kBlockN = kBlockN_;
    static constexpr int kBlockK = kBlockK_;
    static constexpr int kStages = kStages_;

    // TiledMMA
    // using mma_op = decltype(
    //     GMMA::ss_op_selector <
    //         Element, Element, Element,
    //         Shape<Int<kBlockM>, Int<kBlockN>, Int<kBlockK>>
    //     >()
    // );
    using mma_op = decltype(
        SM90_64x64x16_F16F16F16_SS<GMMA::Major::K,GMMA::Major::K>{}
    );
    using mma_traits = MMA_Traits<mma_op>;
    using mma_atom = MMA_Atom<mma_traits>;
    
    // thread repetition
    static constexpr int kMmaEURepeatM = kConsumerWGs;
    static constexpr int kMmaEURepeatN = 1;
    static constexpr int kMmaEURepeatK = 1;

    // thread workload repetition
    using mma_atom_shape = mma_traits::Shape_MNK;
    static constexpr int MmaVM = 1 * kMmaEURepeatM * get<0>(mma_atom_shape{});
    static constexpr int MmaVN = 1 * kMmaEURepeatN * get<1>(mma_atom_shape{});
    static constexpr int MmaVK = 1 * kMmaEURepeatK * get<2>(mma_atom_shape{});
    // this is for problem shape (64x1x1) x (64x1x1) x (16x1x1) = 64x64x16
    
    // Thread repetition 1x1x1 --> 128 threads
    using MMA_EU_RepeatT = decltype(
        make_layout(
            make_shape(Int<kMmaEURepeatM>{}, Int<kMmaEURepeatN>{}, Int<kMmaEURepeatK>{})
        )
    );

    // Thread workload repetition 1x1x1
    // Each mode of this shape can be a layout to do permutation on the corresponding layout mode
    using MMA_V_RepeatT = decltype(
        make_shape(Int<MmaVM>{}, Int<MmaVN>{}, Int<MmaVK>{})
    );

    using TiledMMA = decltype(
        make_tiled_mma(
            mma_atom{},
            MMA_EU_RepeatT{},
            MMA_V_RepeatT{}
        )
    );

    // Shared memory layout
    // Start from Layout Atom, e.g., Layout_K_SW128_Atom
    // K: leading dimension is K, M: leading dimension is N
    // SW128 float: Sw<3,4,3> o smem_ptr[32b](unset) o (_8,_32):(_32,_1)
    // SW64  float: Sw<2,4,3> o smem_ptr[32b](unset) o (_8,_16):(_16,_1)
    // SW32  float: Sw<1,4,3> o smem_ptr[32b](unset) o (_8,_8):(_8,_1)
    // SW128 half:  Sw<3,4,3> o smem_ptr[16b](unset) o (_8,_64):(_64,_1)
    // SW64  half:  Sw<2,4,3> o smem_ptr[16b](unset) o (_8,_32):(_32,_1)
    // SW32  half:  Sw<1,4,3> o smem_ptr[16b](unset) o (_8,_16):(_16,_1)

    // 32b here means 32 bits = 4 bytes = 1 float
    // 16b here means 16 bits = 2 bytes = 1 half

    // One thing to be careful is the Swizzle config B,M,S
    // For example with SW64 half
    // print would display:         Sw<2,4,3> o smem_ptr[16b](unset) o (_8,_32):(_32,_1)
    // print_layout would display:  Sw<2,3,3> o _0 o (_8,_32):(_32,_1)
    // The difference is the above config 2^M is the number of bytes, not the number of halfs
    // 2^4 bytes = 2^3 halfs
    // Similarly, see the layout for SW128 float
    // print displays:              Sw<3,4,3> o smem_ptr[32b](unset) o (_8,_32):(_32,_1)
    // print_layout displays:       Sw<3,2,3> o _0 o (_8,_32):(_32,_1)
    // 2^4 bytes = 2^2 floats
    // From this we know that these builtin swizzle configs always treat consecutive 16 bytes (8 halfs or 4 floats) as one unit (bc M = 4 always)
    // And it also views 1 row as of having 8 units (bc S = 3)

    // Now let's see how we have that swizzle layout
    // Take SW64 half as an example
    // Why is it SW<2,4,3> and (_8,_32):(_32,_1)?
    // Here 64 bytes is the swizzle width:
    // 64 bytes = 4 x 16 bytes
    // This will do swizzle on 4 consecutive 16-byte segments, or 4 consecutive of (8 halfs), which is 32 halfs, which is exactly the width of the atom
    // So why is the number or rows is 8?
    // This comes from the number of physical rows of the swizzle config
    // SW<2,4,3> means 2^2 rows, or 4 rows. Why 4?
    // Bc the pattern repeats every 4 rows, with row n having the same pattern as row (n-4).
    // --> the number of physical rows = 4, and the number of logical rows = 8

    // Read the appendix C.4,5,6 from this paper: https://arxiv.org/pdf/2410.20399
    // to see the 32-byte, 64-byte, and 128-byte swizzle for halfs (you can view it as float by combining 2 halfs as 1 float)

    using SmemLayoutAtom = GMMA::Layout_K_SW128_Atom<T>;

    using SmemLayoutA = decltype(
        tile_to_shape(
            SmemLayoutAtom{},
            make_shape(Int<kBlockM>{}, Int<kBlockK>{}, Int<kStages>{})
        )
    );

    using SmemLayoutB = decltype(
        tile_to_shape(
            SmemLayoutAtom{},
            make_shape(Int<kBlockN>{}, Int<kBlockK>{}, Int<kStages>{})
        )
    );

    // SharedStorage
    using SharedStorage = SharedStorage<T, kStages, SmemLayoutA, SmemLayoutB>;

    // smem_size
    static constexpr int smem_size = sizeof(SharedStorage);

    // MainloopPipeline
    using MainloopPipeline = cutlass::PipelineTmaAsync<kStages>;
};

// collective mainloop
template <typename Kernel_traits>
struct CollectiveMainloop
{
    // 1. extract Kernel_traits
    using Element = typename Kernel_traits::Element;

    static constexpr int kBlockM = Kernel_traits::kBlockM;
    static constexpr int kBlockN = Kernel_traits::kBlockN;
    static constexpr int kBlockK = Kernel_traits::kBlockK;

    using SmemLayoutA = typename Kernel_traits::SmemLayoutA;
    using SmemLayoutB = typename Kernel_traits::SmemLayoutB;

    using TiledMMA = typename Kernel_traits::TiledMMA;

    // 2. decltype of TMA desc
    using ShapeT = Shape<int32_t, int32_t>;
    using StrideT = Shape<int32_t, _1>;
    // using StrideCT = Shape<_1, int32_t>;
    using StrideCT = Shape<int32_t, _1>;
    using LayoutT = Layout<ShapeT, StrideT>;
    using LayoutCT = Layout<ShapeT, StrideCT>;

    using TmaLoadA = decltype(
        make_tma_copy(
            SM90_TMA_LOAD{},
            make_tensor(
                make_gmem_ptr(static_cast<Element const*>(nullptr)),
                ShapeT{},
                StrideT{}
            ),
            SmemLayoutA{}(_, _, 0)
        )
    );
    using TmaLoadB = decltype(
        make_tma_copy(
            SM90_TMA_LOAD{},
            make_tensor(
                make_gmem_ptr(static_cast<Element const*>(nullptr)),
                ShapeT{},
                StrideT{}
            ),
            SmemLayoutB{}(_, _, 0)
        )
    );

    // 3. set the TMA transaction bytes
    static constexpr int kTmaTransactionBytesA = sizeof(ArrayEngine<Element, size(SmemLayoutA{}(_, _, 0))>);
    static constexpr int kTmaTransactionBytesB = sizeof(ArrayEngine<Element, size(SmemLayoutB{}(_, _, 0))>);
    static constexpr int kTmaTransactionBytes = kTmaTransactionBytesA + kTmaTransactionBytesB; // must be a multiple of 16

    // 4. setup the TMA desc (which replace the need of Params in hopper_00)
    // Host-side kernel arguments
    struct Arguments
    {
        Element const* A;
        Element const* B;
        Element *C;
        LayoutT gmemLayoutA;
        LayoutT gmemLayoutB;
        LayoutCT gmemLayoutC;
    };

    // Device-side kernel params
    struct Params
    {
        LayoutT gmemLayoutA;
        LayoutT gmemLayoutB;
        LayoutCT gmemLayoutC;
        TmaLoadA tma_load_A;
        TmaLoadB tma_load_B;
        Element *C;
    };

    static Params
    to_underlying_arguments(const Arguments& args)
    {
        Tensor mA = make_tensor(
            make_gmem_ptr(args.A),
            args.gmemLayoutA
        );

        Tensor mB = make_tensor(
            make_gmem_ptr(args.B),
            args.gmemLayoutB
        );

        TmaLoadA tma_load_A = make_tma_copy(
            SM90_TMA_LOAD{},
            mA,
            SmemLayoutA{}(_, _, 0)
        );
        TmaLoadB tma_load_B = make_tma_copy(
            SM90_TMA_LOAD{},
            mB,
            SmemLayoutB{}(_, _, 0)
        );

        return
        {
            args.gmemLayoutA,
            args.gmemLayoutB,
            args.gmemLayoutC,
            tma_load_A,
            tma_load_B,
            args.C
        };
    }

    // 5. prefetch TMA desc
    CUTLASS_DEVICE
    static void prefetch_tma_descriptors(Params const& mainloop_params)
    {
        cute::prefetch_tma_descriptor(mainloop_params.tma_load_A.get_tma_descriptor());
        cute::prefetch_tma_descriptor(mainloop_params.tma_load_B.get_tma_descriptor());
    }

    // 6. producer
    using MainloopPipeline = typename Kernel_traits::MainloopPipeline;
    using PipelineState = typename MainloopPipeline::PipelineState;

    template <typename SharedStorage>
    CUTLASS_DEVICE
    static void load
    (
        Params const& mainloop_params,
        MainloopPipeline pipeline,
        PipelineState& write_state,
        SharedStorage& shared_storage,
        int NUM_TILES_K
    )
    {
        // gmem tensors
        Tensor mA = mainloop_params.tma_load_A.get_tma_tensor(mainloop_params.gmemLayoutA.shape());
        Tensor mB = mainloop_params.tma_load_B.get_tma_tensor(mainloop_params.gmemLayoutB.shape());

        // tiling
        auto cta_tiler = make_shape(Int<kBlockM>{}, Int<kBlockN>{}, Int<kBlockK>{});
        auto cta_coord = make_coord(blockIdx.y, blockIdx.x, _);
        Tensor gA = local_tile(
            mA,
            cta_tiler,
            cta_coord,
            Step<_1, X, _1>{}
        );  // kBlockM x kBlockK x NUM_TILES_K
        Tensor gB = local_tile(
            mB,
            cta_tiler,
            cta_coord,
            Step<X, _1, _1>{}
        );  // kBlockN x kBlockK x NUM_TILES_K

        // smem tensors
        Tensor sA = make_tensor(
            make_smem_ptr(shared_storage.smem_A.data()),
            SmemLayoutA{}
        );  // kBlockM x kBlockK x PIPE
        Tensor sB = make_tensor(
            make_smem_ptr(shared_storage.smem_B.data()),
            SmemLayoutB{}
        ); // kBlockN x kBlockK x PIPE

        // copy partition
        auto [tAgA, tAsA] = tma_partition(
            mainloop_params.tma_load_A,
            _0{}, Layout<_1>{},
            group_modes<0,2>(sA),
            group_modes<0,2>(gA)
        ); // (TMA, NUM_TILES_K) and (TMA, PIPE)
        auto [tBgB, tBsB] = tma_partition(
            mainloop_params.tma_load_B,
            _0{}, Layout<_1>{},
            group_modes<0,2>(sB),
            group_modes<0,2>(gB)
        ); // (TMA, NUM_TILES_K) and (TMA, PIPE)

        int lane_predicate = cute::elect_one_sync();

        // copy
        if (lane_predicate)
        {
            #pragma unroll 1
            for (int k_tile = 0; k_tile < NUM_TILES_K; ++k_tile)
            {
                pipeline.producer_acquire(write_state);
                // empty_barrier.wait()
                // full_barrier.arrive_and_expect_tx()
                uint64_t* full_barrier = pipeline.producer_get_barrier(write_state);

                auto stage = write_state.index();

                copy(mainloop_params.tma_load_A.with(*full_barrier, 0), tAgA(_, k_tile), tAsA(_, stage));
                copy(mainloop_params.tma_load_B.with(*full_barrier, 0), tBgB(_, k_tile), tBsB(_, stage));

                ++write_state;
            }
        }
    }

    // 7. consumers
    template <typename SharedStorage>
    CUTLASS_DEVICE
    static void mma
    (
        Params const& mainloop_params,
        MainloopPipeline pipeline,
        PipelineState& read_state,
        SharedStorage& shared_storage,
        int NUM_TILES_K
    )
    {
        // gmem tensors
        Tensor mC = make_tensor(
            make_gmem_ptr(mainloop_params.C),
            mainloop_params.gmemLayoutC  
        );
        
        // tiling
        auto cta_tiler = make_shape(Int<kBlockM>{}, Int<kBlockN>{}, Int<kBlockK>{});
        auto cta_coord = make_coord(blockIdx.y, blockIdx.x, _);
        Tensor gC = local_tile(
            mC,
            cta_tiler,
            cta_coord,
            Step<_1, _1, X>{}
        );

        // smem tensors
        Tensor sA = make_tensor(
            make_smem_ptr(shared_storage.smem_A.data()),
            SmemLayoutA{}
        );  // kBlockM x kBlockK x PIPE
        Tensor sB = make_tensor(
            make_smem_ptr(shared_storage.smem_B.data()),
            SmemLayoutB{}
        ); // kBlockN x kBlockK x PIPE

        // partition
        TiledMMA tiled_mma;
        ThrMMA thr_mma = tiled_mma.get_thread_slice(threadIdx.x - cutlass::NumThreadsPerWarpGroup);

        Tensor tCsA = thr_mma.partition_A(sA);  // (MMA,MMA_M,MMA_K, PIPE)
        Tensor tCsB = thr_mma.partition_B(sB);  // (MMA,MMA_N,MMA_K, PIPE)
        Tensor tCgC = thr_mma.partition_C(gC);  // (MMA,MMA_M,MMA_N)
        Tensor tCrC = thr_mma.make_fragment_C(tCgC);  // (MMA,MMA_M,MMA_N)
        clear(tCrC);

        // allocate "fragments"
        // note that in Ampere, the fragments are physically stored in registers
        // but in Hopper, the fragments are not, the tCrA and tCrB are actually the iterators not the data, but still in registers
        Tensor tCrA = thr_mma.make_fragment_A(tCsA);  // iterator (1, MMA_M, MMA_K, PIPE)
        Tensor tCrB = thr_mma.make_fragment_B(tCsB);  // iterator (1, MMA_N, MMA_K, PIPE)

         // MAINLOOP MMA
        #pragma unroll 1
        for (int k_tile = 0; k_tile < NUM_TILES_K; ++k_tile) {
            // Wait for TMA to load this stage of the pipeline
            pipeline.consumer_wait(read_state);
            auto stage = read_state.index();
            warpgroup_arrive();
            // WGMMA with dispatch mode (V,M,K) x (V,N,K) => (V,M,N)
            gemm(tiled_mma, tCrC, tCrA(_,_,_,stage), tCrB(_,_,_,stage), tCrC);
            warpgroup_commit_batch();
        
            // Wait for all MMAs in a K_TILE to complete
            warpgroup_wait<0>();

            // Release the stage of the pipeline for TMA
            pipeline.consumer_release(read_state);
            ++read_state;
        }

        // Make sure all warpgroups have finished mma
        cutlass::arch::NamedBarrier::sync(Kernel_traits::kConsumerWGs * 32 * 4, 0);
        axpby(static_cast<Element>(1.0f), tCrC, static_cast<Element>(0.0f), tCgC);
    }
};

// kernel
template <
    typename Kernel_traits
>
__global__ void cute_hopper_gemm_v01(
    CUTE_GRID_CONSTANT typename CollectiveMainloop<Kernel_traits>::Params const mainloop_params
)
{   
    using CollectiveMainloop = CollectiveMainloop<Kernel_traits>;

    using MainloopPipeline = typename Kernel_traits::MainloopPipeline;
    using PipelineParams = typename MainloopPipeline::Params;
    using PipelineState = typename MainloopPipeline::PipelineState;
    // The Synchronization is orchestrated by the pipeline + pipeline state
    // each thread has its own pipeline state to control the synchronization
    // pipeline state = phase bit + stage + count
    // phase bit is initialized to 0 and flip between 0 and 1
    // stage increments by 1 each time but reset to 0 when it reaches kStages
    // count increments by 1 each time

    // shared memory for data + pipeline
    using SharedStorage = typename Kernel_traits::SharedStorage;
    extern __shared__ char smem_[];
    auto &shared_storage = *reinterpret_cast<SharedStorage*>(smem_);
   
    // Only one thread is elected to perfrom the prefetch
    int warp_idx = cutlass::canonical_warp_idx_sync();
    int lane_predicate = cute::elect_one_sync();

    // prefetch TMA Descriptor
    if (warp_idx == 0 && lane_predicate)
    {
        CollectiveMainloop::prefetch_tma_descriptors(mainloop_params);
    }

    // pipeline initialization
    PipelineParams pipeline_params;

    // set the transaction size
    // Remember that the transaction size is passed to arrive_and_expect_tx of the barrier
    // However, the pipeline.producer_acquire_tx() will do that for us:
    // 1. empty_barrier.wait()
    // 2. full_barrier.arrive_and_expect_tx() (only Hopper does this 2.)
    pipeline_params.transaction_bytes = CollectiveMainloop::kTmaTransactionBytes;
    
    // set the role
    int warp_group_idx = cutlass::canonical_warp_group_idx();    
    pipeline_params.role = warp_group_idx == 0
        ? MainloopPipeline::ThreadCategory::Producer
        : MainloopPipeline::ThreadCategory::Consumer;

    // set the thread leader (scope of warp_group)
    const int warp_group_thread_idx = threadIdx.x % cutlass::NumThreadsPerWarpGroup;
    pipeline_params.is_leader = warp_group_thread_idx == 0;
    pipeline_params.num_consumers = cutlass::NumThreadsPerWarpGroup * Kernel_traits::kConsumerWGs;
    
    MainloopPipeline pipeline(
        shared_storage.pipeline,    // address of the pipeline
        pipeline_params,
        Shape<_1, _1, _1>{}         // todo
    );

    const int NUM_TILES_K = cutlass::ceil_div(
        shape<1>(mainloop_params.gmemLayoutA),
        Kernel_traits::kBlockK
    );

    // We need this to guarantee that the Pipeline init is visible to all producers and consumer blocks in the Cluster
    // This is similar to have the barrier visible to all threads in the Cluster
    cluster_sync();

    // Producer
    if (warp_group_idx == 0)
    {
        cutlass::arch::warpgroup_reg_dealloc<24>();
        int warp_idx_in_warpgroup = __shfl_sync(
            0xffffffff,                     // mask: all lanes in the warp
            (threadIdx.x / 32) % 4,         // value: warp_idx_in_warpgroup
            0                               // src lane
        );

        // only the first warp in the warp group will do the load
        // and only the elected thread inside the first warp will do the load
        if (warp_idx_in_warpgroup == 0)
        {
            PipelineState write_state = cutlass::make_producer_start_state<MainloopPipeline>();
            // phase bit = 1, stage = 0, count = 0
            // 1. the phase bit of write_state is to control the empty_barrier (sounds counterintuitive):
            // pipeline.producer_acquire(write_state) or empty_barrier.wait(): when all threads have arrived, the phase bit will be flipped to 1
            // 2. the stage of write_state is to control the full_barrier:
            // pipeline.producer_get_barrier(write_state): it gets the full_barrier_ptr_[stage]

            CollectiveMainloop::load(
                mainloop_params,
                pipeline,
                write_state,
                shared_storage,
                NUM_TILES_K
            );
        }
    }
    else    // Consumer
    {
        cutlass::arch::warpgroup_reg_alloc<240>();
        PipelineState read_state;

        CollectiveMainloop::mma(
            mainloop_params,
            pipeline,
            read_state,
            shared_storage,
            NUM_TILES_K
        );
    }
}

// launch
template<typename T>
void launch_cute_hopper_gemm_kernel_v01(
    size_t m, size_t n, size_t k,
    const T *alpha,
    const T *A, size_t lda,
    const T *B, size_t ldb,
    const T *beta,
    T *C, size_t ldc,
    hipStream_t stream
)
{   
    // Block shape and cta tiler
    constexpr int kWarps_ = 12;
    constexpr int kBlockM_ = 256;
    constexpr int kBlockN_ = 128;
    constexpr int kBlockK_ = 128;
    constexpr int kStages_ = 2;

    using Kernel_traits = KernelTraits<T, kWarps_,kBlockM_, kBlockN_, kBlockK_, kStages_>;

    using SmemLayoutA = typename Kernel_traits::SmemLayoutA;
    using SmemLayoutB = typename Kernel_traits::SmemLayoutB;
    using TiledMMA = typename Kernel_traits::TiledMMA;

    // setup TMA desc like v00 but using CollectiveMainloop
    int M = int(m); int N = int(n); int K = int(k);
    auto gmemLayoutA = make_layout(make_shape(M, K), make_stride(K, _1{}));
    auto gmemLayoutB = make_layout(make_shape(N, K), make_stride(K, _1{}));
    // auto gmemLayoutC = make_layout(make_shape(M, N), make_stride(_1{}, M));
    auto gmemLayoutC = make_layout(make_shape(M, N), make_stride(N, _1{}));
    using Collective_mainloop = CollectiveMainloop<Kernel_traits>;
    typename Collective_mainloop::Params mainloop_params = Collective_mainloop::to_underlying_arguments(
        {
            A, B, C,
            gmemLayoutA, gmemLayoutB, gmemLayoutC
        }
    );

    // Launch parameter setup
    constexpr int smem_size = Kernel_traits::smem_size;
    dim3 block{Kernel_traits::kThreads, 1U, 1U};
    dim3 cluster{1, 1, 1};
    dim3 grid{
        cute::size(ceil_div(n, kBlockN_)),
        cute::size(ceil_div(m, kBlockM_)),
        1U
    };
    cutlass::ClusterLaunchParams launch_params{grid, block, cluster, smem_size, stream};

    void const* kernel = reinterpret_cast<void const*>(&cute_hopper_gemm_v01 <Kernel_traits>);

    if (smem_size >= 48 * 1024) // 48KB
    {
        CUTE_CHECK_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));
    }

    // kernel launch
    cutlass::Status status = cutlass::launch_kernel_on_cluster(
        launch_params,
        kernel,
        mainloop_params
    );
    CUTE_CHECK_LAST();

    if (status != cutlass::Status::kSuccess)
    {
        std::cerr << "Kernel launch failed with status: " << std::endl;
    }

}

// explicit instantiation                      
template void launch_cute_hopper_gemm_kernel_v01<cute::half_t>(size_t m, size_t n, size_t k,
                                    const cute::half_t *alpha,
                                    const cute::half_t *A, size_t lda,
                                    const cute::half_t *B, size_t ldb,
                                    const cute::half_t *beta,
                                    cute::half_t *C, size_t ldc,
                                    hipStream_t stream);                                    